#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <builtin_types.h>
#include "gpu.h"

namespace NyxusGpu
{
	bool gpu_delete(void* devptr)
	{
		CHECKERR(hipFree(devptr));
		return true;
	}

	bool allocate_on_device(void** ptr, size_t szb)
	{
		CHECKERR(hipMalloc(ptr, szb));
		return true;
	}

	bool upload_on_device (void* devbuffer, void* hobuffer, size_t szb)
	{
		CHECKERR(hipMemcpy(devbuffer, hobuffer, szb, hipMemcpyHostToDevice));
		return true;
	}

	bool download_on_host (void* hobuffer, void* devbuffer, size_t szb)
	{
		CHECKERR(hipMemcpy(hobuffer, devbuffer, szb, hipMemcpyDeviceToHost));
		return true;
	}

	bool devicereduce_evaluate_buffer_szb (size_t & devicereduce_buf_szb, size_t maxLen)
	{
		auto ercode = hipcub::DeviceReduce::Sum ((void*)nullptr, devicereduce_buf_szb, (double*)nullptr /*in: d_prereduce*/, (double*)nullptr /*out: result*/, maxLen /*in: cloudlen*/);
		CHECKERR(ercode);
		return true;
	}

	bool gpu_get_free_mem(size_t& amt)
	{
		size_t mf, ma;
		CHECKERR(hipMemGetInfo(&mf, &ma));
		amt = mf;
		return true;
	}

}

